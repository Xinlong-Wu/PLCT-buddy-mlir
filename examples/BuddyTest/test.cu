#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void addmm_kernel(float* mat1, float* mat2, float* bias, float* result, int m, int n, int k) {
    for (int i = threadIdx.x; i < m; i += blockDim.x) {
        for (int j = threadIdx.y; j < n; j += blockDim.y) {
            float sum = 0.0f;

            // Mat mul
            for (int l = 0; l < k; l++) {
                sum += mat1[i * k + l] * mat2[l * n + j];
            }

            // add bias
            result[i * n + j] = sum + bias[i];
        }
    }
}

int main() {
    int m = 4; // row
    int n = 4; // col
    int k = 2; // inner dim

    size_t mat1_size = m * k * sizeof(float);
    size_t mat2_size = k * n * sizeof(float);
    size_t bias_size = m * sizeof(float);
    size_t result_size = m * n * sizeof(float);

    // 初始化矩阵（此处省略初始化代码）
    float h_mat1[4 * 2] = {
        0, 9,
        1, 2,
        3, 4,
        5, 6
    };

    float h_mat2[2 * 4] = {
        1, 2, 3, 4,
        1, 2, 3, 4
    };

    float h_bias[4 * 4] = {
        1, 1, 1, 1,
        1, 1, 1, 1,
        1, 1, 1, 1,
        1, 1, 1, 1
    };

    float h_result[4 * 4] = {-1};


    float* d_mat1;
    float* d_mat2;
    float* d_bias;
    float* d_result;

    hipMalloc(&d_mat1, mat1_size);
    hipMalloc(&d_mat2, mat2_size);
    hipMalloc(&d_bias, bias_size);
    hipMalloc(&d_result, result_size);

    hipMemcpy(d_mat1, h_mat1, mat1_size, hipMemcpyHostToDevice);
    hipMemcpy(d_mat2, h_mat2, mat2_size, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, h_bias, bias_size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x, (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    addmm_kernel<<<128, 1, 1>>>(d_mat1, d_mat2, d_bias, d_result, m, n, k);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipMemcpy(h_result, d_result, result_size, hipMemcpyDeviceToHost);

    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f ", h_result[i * n + j]);
        }
        printf("\n");
    }

    // 清理
    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_bias);
    hipFree(d_result);

    return 0;
}
